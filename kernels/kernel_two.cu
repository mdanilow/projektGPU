
#include <hip/hip_runtime.h>

__device__ bool inLeftBorder();
__device__ bool inRightBorder(int imWidth, int BLOCK_WIDTH, int batchNum);
__device__ bool inTopBorder();
__device__ bool inBottomBorder(int imHeight, int BLOCK_WIDTH, int batchNum);
__device__ int globalAddr(const int x, const int y, const int height);
__device__ int findRoot(int equivalenceArray[], int elementAddress);
__device__ void Union(int equivalenceArray[], const int segmentsArray[], const int elementAddress0, const int elementAddress1, int* changedPtr);


__global__ void mergeTiles(
        const int* dSegData,
        int* dLabelsData,
        const int height,
        const int width,
        const int BLOCK_WIDTH){

    __shared__ int changed; //shared memory used to check whether the solution is final or not

    int subBlockY = blockIdx.y*blockDim.y + threadIdx.y;
    int subBlockX = blockIdx.x*blockDim.x + threadIdx.x;

    int x, y = 0;
    int repetitions = BLOCK_WIDTH / blockDim.z; //how many times are the thread reused for the given subblock?
    
    // printf("blockIdx.x: %d\nblockIdx.y: %d\nthreadIdx.x: %d\nthreadIdx.y: %d\nsubBlockX: %d\nsubBlockY: %d", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, subBlockX, subBlockY);
    
    while(1) {

        if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) 
            changed = 0;
        __syncthreads();

        //process the bottomhorizontal border
        for(int i = 0; i < repetitions; i++){
            //pixel coordinates
            x = subBlockX * BLOCK_WIDTH + threadIdx.z + i*blockDim.z;
            y = (subBlockY+1) * BLOCK_WIDTH - 1;

            if(globalAddr(x+1, y+1, height) < 0 || globalAddr(x+1, y+1, height) > width*height-1)
                printf("width: %d, height: %d, horizontal %d\n", width, height, globalAddr(x, y, height));  

            if(!inLeftBorder())
                Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x-1, y+1, height), &changed);

            Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x, y+1, height), &changed);  

            if(!inRightBorder(width, BLOCK_WIDTH, i))
                Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x+1, y+1, height), &changed);
        }

        //process the right vertical border
        for(int i = 0; i < repetitions; i++){
            //pixel coordinates
            y = subBlockY * BLOCK_WIDTH + threadIdx.z + i*blockDim.z;
            x = (subBlockX+1) * BLOCK_WIDTH - 1;

            if(globalAddr(x, y, height) < 0 || globalAddr(x, y, height) > width*height-1)
                printf("vertical %d\n", globalAddr(x, y, height));  

            if(!inTopBorder())
                Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x+1, y-1, height), &changed);

            Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x+1, y, height), &changed);

            if(!inBottomBorder(height, BLOCK_WIDTH, i))
                Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x+1, y+1, height), &changed);
        }

            __syncthreads();
            if(changed == 0) break; //no changes −> the tiles are merged
            __syncthreads();    
    }
}

__device__ bool inLeftBorder(){

    return (threadIdx.x == 0 && blockIdx.x == 0);
}

__device__ bool inRightBorder(int imWidth, const int BLOCK_WIDTH, int batchNum){

    int subBlockX = blockIdx.x*blockDim.x + threadIdx.x;
    int x = subBlockX * BLOCK_WIDTH + threadIdx.z + batchNum*blockDim.z;

    return (x >= imWidth);
}

__device__ bool inTopBorder(){

    return (threadIdx.y == 0 && blockIdx.y == 0);
}

__device__ bool inBottomBorder(int imHeight, const int BLOCK_WIDTH, int batchNum){

    int subBlockY = blockIdx.y*blockDim.y + threadIdx.y;
    int y = subBlockY * BLOCK_WIDTH + threadIdx.z + batchNum*blockDim.z;

    return (y == imHeight);
}

__device__ int globalAddr(const int x, const int y, const int height){
    return x * height + y;
}

__device__ int findRoot(int equivalenceArray[], int elementAddress){

    while(equivalenceArray[elementAddress] != elementAddress)
        elementAddress = equivalenceArray[elementAddress];
    return elementAddress;
}

__device__ void Union(int equivalenceArray[], const int segmentsArray[], const int elementAddress0, const int elementAddress1, int* changedPtr){

    if(segmentsArray[elementAddress0] == segmentsArray[elementAddress1]){

        int root0 = findRoot(equivalenceArray, elementAddress0);
        int root1 = findRoot(equivalenceArray, elementAddress1);
        //connect an equivalence tree with a higher label to the tree with a lower label
        if(root0 < root1){
            atomicMin(equivalenceArray + root1, root0);
            *changedPtr = 1;
        }
        else if(root1 < root0) {
            atomicMin(equivalenceArray + root0, root1);
            *changedPtr = 1;
        }
    }
}
