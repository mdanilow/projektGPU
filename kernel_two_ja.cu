#include "hip/hip_runtime.h"
#define BLOCK_WIDTH 16

__device__ bool inLeftBorder();
__device__ bool inRightBorder();
__device__ bool inTopBorder();
__device__ bool inBottomBorder();
__device__ int globalAddr(const int x, const int y, const int height);
__device__ int findRoot(int equivalenceArray[], int elementAddress);
__device__ void Union(int equivalenceArray[], const int elementAddress0, const int elementAddress1);


__global__ void mergeTiles(
        const double* dSegData,
        int* dLabelsData,
        const int height,
        const int width){

    __shared__ int changed; //shared memory used to check whether the solution is final or not

    int subBlockY = blockIdx.y*blockDim.y + threadIdx.y;
    int subBlockX = blockIdx.x*blockDim.x + threadIdx.x;

    int x, y = 0;

    int repetitions = int(BLOCK_WIDTH/depth); //how many times are the thread reused for the given subblock?
     
    while(1) {
        if(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) 
            changed = 0;
        __syncthreads();

        //process the bottomhorizontal border
        for(int i=0; i < repetitions; i++) {

            //pixel coordinates
            x = subBlockX * BLOCK_WIDTH + threadIdx.z + i*blockDim.z;
            y = (subBlockY+1) * BLOCK_WIDTH - 1;

            if(!inLeftBorder())
                Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x-1, y+1, height), &changed);

            Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x, y+1, height), &changed);  

            if(!inRightBorder(i, width))
                Union(dLabelsData, dSegData, globalAddr(x, y, height), globalAddr(x+1, y+1, height), &changed);
        }

        //process the right vertical border
        for(int i=0; i < repetitions; i++) {

            //pixel coordinates
            y = (subBlockY+1) * BLOCK_WIDTH + threadIdx.z + i*blockDim.z;
            x = (subBlockX+1) * BLOCK_WIDTH - 1;

            if(!inTopBorder())
                Union(dLabelsData, globalAddr(x, y, height), globalAddr(x-1, y-1, height));

            if(!inBottomBorder())
                Union(dLabelsData, globalAddr(x, y, height), globalAddr(x-1, y-1, height));
        }
        __syncthreads();
        if(changed == 0) break; //no changes −> the tiles are merged
        __syncthreads();
    }
}

__device__ bool inLeftBorder(){

    return (threadIdx.x == 0 && blockIdx.x == 0);
}

__device__ bool inRightBorder(int portionNum, int imWidth){

    x = subBlockX * BLOCK_WIDTH + threadIdx.z + portionNum*blockDim.z;

    return (x == imWidth);
}

__device__ bool inTopBorder(){

    return (threadIdx.y == 0 && blockIdx.y == 0);
}

__device__ bool inBottomBorder(){

    return (blockIdx.y == (blockDim.y - 1) && threadIdx.y == BLOCK_HEIGHT-1);
}

__device__ int globalAddr(const int x, const int y, const int height){
    return x * height + y;
}

__device__ int findRoot(int equivalenceArray[], int elementAddress){

    while(equivalenceArray[elementAddress] != elementAddress)
        elementAddress = equivalenceArray[elementAddress];
    return elementAddress;
}

__device__ void Union(int equivalenceArray[], int segmentsArray[], const int elementAddress0, const int elementAddress1, int* changedPtr){

    if(segmentsArray[elementAddress0] == segmentsArray[elementAddress1]){

        int root0 = findRoot(equivalenceArray, elementAddress0);
        int root1 = findRoot(equivalenceArray, elementAddress1);
        //connect an equivalence tree with a higher label to the tree with a lower label
        if(root0 < root1){
            atomicMin(equivalenceArray + root1, root0);
            *changedPtr = 1;
        }
        else if(root1 < root0) {
            atomicMin(equivalenceArray + root0, root1);
            *changedPtr = 1;
        }
    }
}
