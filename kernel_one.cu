
#include <hip/hip_runtime.h>
#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16

enum NEIGH_TYPE {NEIGH_FOUR = 0, NEIGH_EIGHT = 1};

__device__ int getNeighboursLocalIndexes(int neighbours[], int nType);
__device__ int getLocalIndex(int localRow, int localCol);
__device__ bool inLocalBorder();
__device__ int findRoot(int equivalenceMatrix[], int elementIndex);


__global__ void localCCL(const double* input, int* output, const int height, const int width){

    __shared__ int segments[BLOCK_WIDTH * BLOCK_HEIGHT];
    __shared__ int labels[BLOCK_WIDTH * BLOCK_HEIGHT];
    __shared__ int changed;
    
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int localRow = threadIdx.y;
    int localCol = threadIdx.x;
    int localIndex = localRow * blockDim.y + localCol;
    int globalIndex = col * height + row;
    int newLabel;
    int nType = NEIGH_EIGHT;

    // load corresponding image tile to shared memory
    segments[localIndex] = input[globalIndex];

    // clear borders in every tile
    // if(inLocalBorder()){
    //     segments[localIndex] = 0;
    // }

    __syncthreads();
    int label = localIndex;
    int neighboursIndexes[8];
    int numOfNeighbours;

    if(!inLocalBorder()){

        while(1){

            labels[localIndex] = label;

            if(localRow == 0 && localCol == 0)
                changed = 0;

            __syncthreads();
            newLabel = label;

            numOfNeighbours = getNeighboursLocalIndexes(neighboursIndexes, nType);
            
            for(int n = 0; n < numOfNeighbours; n++)
                if(segments[localIndex] == segments[neighboursIndexes[n]])
                    newLabel = min(newLabel, labels[neighboursIndexes[n]]);

            __syncthreads();

            if(newLabel < label){

                atomicMin(labels + label, newLabel);
                changed = 1;
            }

            __syncthreads();

            if(changed == 1)
                break;
        }
    }

    if(col < width && row < height)
        output[globalIndex] = segments[localIndex];
}


//returns length of neighbours list
__device__ int getNeighboursLocalIndexes(int neighbours[], int nType){

    int localRow = threadIdx.x;
    int localCol = threadIdx.y;
    int length;

    if(nType == NEIGH_FOUR){
        
        neighbours[0] = getLocalIndex(localRow - 1, localCol);
        neighbours[1] = getLocalIndex(localRow, localCol + 1);
        neighbours[2] = getLocalIndex(localRow + 1, localCol);
        neighbours[3] = getLocalIndex(localRow, localCol - 1);

        length = 4;
    }

    else if(nType == NEIGH_EIGHT){

        neighbours[0] = getLocalIndex(localRow - 1, localCol - 1);
        neighbours[1] = getLocalIndex(localRow - 1, localCol);
        neighbours[2] = getLocalIndex(localRow - 1, localCol + 1);
        neighbours[3] = getLocalIndex(localRow, localCol + 1);
        neighbours[4] = getLocalIndex(localRow + 1, localCol + 1);
        neighbours[5] = getLocalIndex(localRow + 1, localCol);
        neighbours[6] = getLocalIndex(localRow + 1, localCol - 1);
        neighbours[7] = getLocalIndex(localRow, localCol - 1);

        length = 8;
    }

    return length;
}


__device__ int getLocalIndex(int localRow, int localCol){

    return localCol * blockDim.y + localRow;
}


__device__ bool inLocalBorder(){

    return (threadIdx.x == 0 || threadIdx.x == BLOCK_WIDTH-1 || threadIdx.y == 0 || threadIdx.y == BLOCK_HEIGHT-1);
}


__device__ int findRoot(int equivalenceMatrix[], int elementIndex){

    if(inLocalBorder())
        equivalenceMatrix[elementIndex] = 255;
    return 0;
}